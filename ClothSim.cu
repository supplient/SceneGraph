#include "hip/hip_runtime.h"
#include <DirectXMath.h>

#include "hip/hip_vector_types.h"
#include "hip/hip_runtime_api.h"
#include ""

#include "SimulationConst.h"
#include "CudaInterface.h"

using namespace sim;
using DirectX::XMFLOAT3;


namespace sim {
	__managed__ XMFLOAT3* x = nullptr;
	__managed__ size_t x_pitch;
	__managed__ XMFLOAT3* v = nullptr;
	__managed__ size_t v_pitch;
	__managed__ float* w = nullptr;
	__managed__ size_t w_pitch;

	__managed__ XMFLOAT3* x_next = nullptr;
	__managed__ XMFLOAT3* v_next = nullptr;

	__constant__ float3 windForce = { 2.4f, 0.0f, 2.4f };
	constexpr float k_tension = 25.0f;
	constexpr float d_tension = 2.0f;
	constexpr float k_cut = k_tension/10.0f;
	constexpr float d_cut = d_tension/10.0f;
	constexpr float k_bend = k_cut/10.0f;
	constexpr float d_bend = d_cut/10.0f;
	constexpr float k_drag = 1.0f;
	constexpr float k_rise = 0.5f;

	auto max = [](int a, int b) {return a > b ? a : b; };
	dim3 block(16, 16, 1);
	dim3 grid(max((n-1)/16+1, 1), max((n-1)/16+1, 1), 1);
}


__device__ int CalVertID(int i, int j, bool isFront)
{
	if (isFront)
		return i * n + j;
	else
		return i * n + j + n * n;
}

__device__ unsigned int GetThreadi()
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ unsigned int GetThreadj()
{
	return blockIdx.y * blockDim.y + threadIdx.y;
}

__device__ void* GetTwoDimArray(void* arr, unsigned int i, unsigned int j, size_t pitch, size_t eleSize)
{
	return (void*)((char*)arr + i * pitch + j * eleSize);
}

__device__ float3& GetX(int i, int j)
{
	return *(float3*)GetTwoDimArray(x, i, j, x_pitch, sizeof(float3));
}

__device__ float3& GetV(int i, int j)
{
	return *(float3*)GetTwoDimArray(v, i, j, v_pitch, sizeof(float3));
}

__device__ float3& GetXNext(int i, int j)
{
	return *(float3*)GetTwoDimArray(x_next, i, j, x_pitch, sizeof(float3));
}

__device__ float3& GetVNext(int i, int j)
{
	return *(float3*)GetTwoDimArray(v_next, i, j, v_pitch, sizeof(float3));
}

__device__ float& GetW(int i, int j)
{
	return *(float*)GetTwoDimArray(w, i, j, w_pitch, sizeof(float));
}

__device__ float3 CalForceFromNeighbor(int i, int j, int a, int b, float k, float d, float l0)
{
	float3 F = { 0, 0, 0 };
	if (a >= 0 && a < n && b >= 0 && b < n) {
		float3 src = GetX(a, b);
		float3 dst = GetX(i, j);
		float3 xDelta = src - dst;
		float xDeltaLen = length(xDelta);
		xDelta = normalize(xDelta);
		float3 vSrc = GetV(a, b);
		float3 vDst = GetV(i, j);
		float3 vDelta = vSrc - vDst;
		float lenDiff = xDeltaLen - l0;

		F += k * lenDiff * xDelta; // ��������
		F += d * dot(xDelta, vDelta) * xDelta; // ��������
	}
	return F;
}

__device__ float3 CalWindForceFromNeighbor(int i, int j, int a, int b)
{
	float3 F = { 0.0f, 0.0f, 0.0f };
	if (a >= 0 && a < n - 1 && b >= 0 && b < n - 1) {
		float3 vLeftUp		= GetV(a  , b  );
		float3 vLeftDown	= GetV(a+1, b  );
		float3 vRightUp		= GetV(a  , b+1);
		float3 vRightDown	= GetV(a+1, b+1);
		float3 v_quad = vLeftUp + vLeftDown + vRightUp + vRightDown;
		v_quad /= 4.0f;
		float3 v_rel = v_quad - windForce;
		float3 norm = cross(GetX(a, b) - GetX(a + 1, b), GetX(a, b) - GetX(a, b + 1));
		norm = normalize(norm);
		float S = length(cross(GetX(a, b) - GetX(a + 1, b), GetX(a, b) - GetX(a, b + 1)));
		S += length(cross(GetX(a + 1, b + 1) - GetX(a + 1, b), GetX(a + 1, b + 1) - GetX(a, b + 1)));
		S /= 2.0f;
		S *= abs(dot(norm, v_rel));

		F += -0.25 * S * k_drag * v_rel;
        F += - 0.25 * S * k_rise * cross(v_rel, normalize(cross(norm, v_rel)));
	}
	return F;
}

__global__ void kernel_ClothSimulation(Vertex* verts, float time)
{
	int i = GetThreadi();
	int j = GetThreadj();

	if (i < n && j < n) {
		// Cal Force
		float3 F = { 0.0f, 0.0f, 0.0f };
		{
			// ����
			F += CalForceFromNeighbor(i, j, i - 1, j, k_tension, d_tension, lConst);
			F += CalForceFromNeighbor(i, j, i + 1, j, k_tension, d_tension, lConst);
			F += CalForceFromNeighbor(i, j, i, j-1, k_tension, d_tension, lConst);
			F += CalForceFromNeighbor(i, j, i, j+1, k_tension, d_tension, lConst);
			// ƽ���ڼ�����
			F += CalForceFromNeighbor(i, j, i-1, j-1, k_cut, d_cut, lCutConst);
			F += CalForceFromNeighbor(i, j, i+1, j-1, k_cut, d_cut, lCutConst);
			F += CalForceFromNeighbor(i, j, i-1, j+1, k_cut, d_cut, lCutConst);
			F += CalForceFromNeighbor(i, j, i+1, j+1, k_cut, d_cut, lCutConst);
			// ƽ����������
			F += CalForceFromNeighbor(i, j, i-2, j, k_bend, d_bend, lBendConst);
			F += CalForceFromNeighbor(i, j, i+2, j, k_bend, d_bend, lBendConst);
			F += CalForceFromNeighbor(i, j, i, j-2, k_bend, d_bend, lBendConst);
			F += CalForceFromNeighbor(i, j, i, j+2, k_bend, d_bend, lBendConst);
			// ����
			F += CalWindForceFromNeighbor(i, j, i - 1, j - 1);
			F += CalWindForceFromNeighbor(i, j, i-1, j  );
			F += CalWindForceFromNeighbor(i, j, i  , j-1);
			F += CalWindForceFromNeighbor(i, j, i  , j  );
		}

		// Cal v_next & x_next
		{
			float3& v_next_ele = GetVNext(i, j);
			float& w_ele = GetW(i, j);
			v_next_ele = GetV(i, j) + w_ele * F * timestep;

			float3& x_next_ele = GetXNext(i, j);
			x_next_ele = GetX(i, j) + v_next_ele * timestep;
		}

		// Switch now and next
		{
			GetX(i, j) = GetXNext(i, j);
			GetV(i, j) = GetVNext(i, j);
		}

		// Update geometry's actual vertex position
		{
			// Just a copy.
			float3& x_ele = GetX(i, j);
			Vertex& vert = verts[CalVertID(i, j, true)];
			vert.pos.x = x_ele.x;
			vert.pos.y = x_ele.y;
			vert.pos.z = x_ele.z;

			// Not forget the back face
			verts[CalVertID(i, j, false)].pos.x = vert.pos.x;
			verts[CalVertID(i, j, false)].pos.y = vert.pos.y;
			verts[CalVertID(i, j, false)].pos.z = vert.pos.z;
		}
	}
}

__device__ void CalNormalWithNeighbor(
	float3& out, uint& count,
	int i, int j,
	int ui, int uj,
	int vi, int vj
)
{
	if (
		ui >= 0 && ui < n && uj >= 0 && uj < n &&
		vi >= 0 && vi < n && vj >= 0 && vj < n
		) {
		out += normalize(cross(
			GetX(ui, uj) - GetX(i, j),
			GetX(vi, vj) - GetX(i, j)
		));
		count += 1;
	}
}

__global__ void kernel_UpdateNormal(Vertex* verts)
{
	int i = GetThreadi();
	int j = GetThreadj();

	if (i < n && j < n) {
		float3 norm = { 0.0f, 0.0f, 0.0f };
		uint count = 0;
		CalNormalWithNeighbor(norm, count, i, j, i - 1, j, i, j + 1);
		CalNormalWithNeighbor(norm, count, i, j, i, j + 1, i + 1, j);
		CalNormalWithNeighbor(norm, count, i, j, i + 1, j, i, j - 1);
		CalNormalWithNeighbor(norm, count, i, j, i, j - 1, i - 1, j);
		norm /= float(count);

		Vertex& vert = verts[CalVertID(i, j, true)];
		vert.normal.x = norm.x;
		vert.normal.y = norm.y;
		vert.normal.z = norm.z;

		// Not forget the back face
		Vertex& backVert = verts[CalVertID(i, j, false)];
		backVert.normal.x = -norm.x;
		backVert.normal.y = -norm.y;
		backVert.normal.z = -norm.z;
	}
}

__global__ void kernel_Init()
{
	auto i = GetThreadi();
	auto j = GetThreadj();

	if (i < n && j < n) {
		float3& x_ele = *(float3*)GetTwoDimArray(x, i, j, x_pitch, sizeof(float3));
		x_ele.x = (i - (n - 1) / 2.0f) * lConst;
		x_ele.y = (j - (n - 1) / 2.0f) * lConst;
		x_ele.z = 0.0f;

		float3& v_ele = *(float3*)GetTwoDimArray(v, i, j, v_pitch, sizeof(float3));
		v_ele.x = 0;
		v_ele.y = 0;
		v_ele.z = 0;

		float& w_ele = *(float*)GetTwoDimArray(w, i, j, w_pitch, sizeof(float));
		if (i == 0)
			w_ele = 0;
		else
			w_ele = 1 / m;
	}
}

namespace sim {
	void InitClothSimulation(hipStream_t streamToRun)
	{
		// Alloc device memory
		hipMallocPitch(&x, &x_pitch, n * sizeof(XMFLOAT3), n);
		hipMallocPitch(&v, &v_pitch, n * sizeof(XMFLOAT3), n);
		hipMallocPitch(&w, &w_pitch, n * sizeof(float), n);

		hipMallocPitch(&x_next, &x_pitch, n * sizeof(XMFLOAT3), n);
		hipMallocPitch(&v_next, &v_pitch, n * sizeof(XMFLOAT3), n);

		// Initialize
		kernel_Init<<<grid, block, 0, streamToRun >>>();

		getLastCudaError("kernel_Init execution failed.\n");
	}

	void ClothSimulation(Vertex* verts, hipStream_t streamToRun, float time)
	{

		kernel_ClothSimulation<<<grid, block, 0, streamToRun >>>(
			verts, time
		);

		getLastCudaError("kernel_ClothSimulation execution failed.\n");
	}

	void UpdateNormal(Vertex* verts, hipStream_t streamToRun)
	{
		kernel_UpdateNormal<<<grid, block, 0, streamToRun >>> (
			verts
		);
		getLastCudaError("kernel_UpdateNormal execution failed.\n");
	}

	void FreeClothSimulation(hipStream_t streamToRun)
	{
		// TODO
	}
}