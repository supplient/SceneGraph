#include "hip/hip_runtime.h"
#include "CudaInterface.h"
#include "SimulationConst.h"
#include "hip/hip_runtime_api.h"
#include ""

using namespace sim;

__device__ int CalVertID(int i, int j, bool isFront)
{
	if (isFront)
		return i * n + j;
	else
		return i * n + j + n * n;
}

__global__ void kernel_ClothSimulation(Vertex* verts, float time)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

	Vertex& vert = verts[CalVertID(i, j, true)];
	vert.pos.z = sin(time);

	verts[CalVertID(i, j, false)].pos.z = vert.pos.z;
}

void ClothSimulation(Vertex* verts, hipStream_t streamToRun, float time)
{
	auto max = [](int a, int b) {return a > b ? a : b; };

	dim3 block(16, 16, 1);
	dim3 grid(max(n / 16, 1), max(n / 16, 1), 1);

	kernel_ClothSimulation<<<grid, block, 0, streamToRun >>>(
		verts, time
	);

	getLastCudaError("kernel_ClothSimulation execution failed.\n");
}